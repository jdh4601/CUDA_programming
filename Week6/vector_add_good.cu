#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int N = 1000000;
    size_t size = N * sizeof(float);
    
    printf("=== 좋은 버전 (최적화됨) ===\n");
    printf("벡터 크기: %d 원소\n\n", N);
    
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    for (int iter = 0; iter < 10; iter++) {
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    }
    
    hipDeviceSynchronize();
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    printf("%.1f, %.1f, %.1f\n", h_A[999999], h_B[999999], h_C[999999]);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
