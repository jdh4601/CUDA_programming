#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1 << 20;  // 1M elements
    const int bytes = N * sizeof(float);

    // Host 메모리
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // device 메모리
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    
    // 동기
    // 1. H -> D 복사. CPU 쓰레드는 대기
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    // 2. 커널 런치는 비동기
    vectorAdd<<<grid, block>>>(d_A, d_B, d_C, N);
    // 3. 커널 완료 대기
    hipDeviceSynchronize(); 
    // 4. D -> H 복사
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost); // host로 복사
    printf("결과: c[0] = %f\n", h_C[0]);

    // 비동기
    hipStream_t s;
    hipStreamCreate(&s); // stream 생성

    hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice, s); // 복사를 stream에 넣는다.
    hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice, s);

    vectorAdd<<<grid, block, 0, s>>>(d_A, d_B, d_C, N);

    hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost, s);
    
    hipStreamSynchronize(s); // stream에 넣은 모든 작업 끝날때까지 스레드 대기
    printf("비동기 결과: c[0] = %f\n", h_C[0]);

    hipStreamDestroy(s);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}