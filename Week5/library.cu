
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

int main() {
    hipSetDevice(4); // gpu 지정하기
    const int size = 20;
    // create number generator
    hiprandGenerator_t generator;
    hiprandStatus_t status = hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    if (status != HIPRAND_STATUS_SUCCESS) {
        printf("failed to create generator\n");
        return 1;
    }

    // set the seed
    status = hiprandSetPseudoRandomGeneratorSeed(generator, 1234);
    if (status != HIPRAND_STATUS_SUCCESS) {
        printf("Failed to set generator seed\n");
        hiprandDestroyGenerator(generator);
        return 1;
    }

    // allocate memory on the host
    unsigned int* hostArray = new unsigned int[size];

    status = hiprandGenerate(generator, hostArray, size);
    if (status != HIPRAND_STATUS_SUCCESS) {
        printf("failed to generate random num");
        delete[] hostArray;
        hiprandDestroyGenerator(generator);
        return 1;
    }

    // print random number
    for (int i = 0; i < size; i++) {
        printf("%u\n", hostArray[i]);
    }
    
    // clean up
    delete[] hostArray;
    hiprandDestroyGenerator(generator);

    return 0;
}