#include <stdio.h>
#include <hip/hip_runtime.h>

// device 전역 변수 -> local에 위치 x
__device__ int d_x;
__device__ int d_y;
__device__ int a;
__device__ int b;

// kernel
__global__ void addKernel(int x, int y, int* result) {
    *result = x + y;
}

__global__ void multiplyKernel(int a, int b, int* result) {
    *result = a * b;
}

int main() {
    int h_x = 2;
    int h_y = 4;    
    int *d_result_add, *d_result_multiply;
    int *h_add, *h_mul;

    hipHostMalloc(&h_add, sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_mul, sizeof(int), hipHostMallocDefault);

    hipStream_t s0, s1; // 각 디바이스별 스트림 생성

    // GPU device 1
    hipSetDevice(0); // GPU 지정하기
    hipStreamCreate(&s0); // stream 만들기
    hipMalloc((void**)&d_result_add, sizeof(int)); // 4byte 메모리 할당
    addKernel<<<1, 1, 0, s0>>>(h_x, h_y, d_result_add);
    hipMemcpyAsync(h_add, d_result_add, sizeof(int), hipMemcpyDeviceToHost, s0);

    // GPU device 2
    hipSetDevice(1);
    hipStreamCreate(&s1);
    hipMalloc((void**)&d_result_multiply, sizeof(int));
    multiplyKernel<<<1, 1, 0, s1>>>(h_x, h_y, d_result_multiply);
    hipMemcpyAsync(h_mul, d_result_multiply, sizeof(int), hipMemcpyDeviceToHost, s1);

    // 마지막에만 기다림
    hipSetDevice(0); hipStreamSynchronize(s0);
    hipSetDevice(1); hipStreamSynchronize(s1);

    printf("result_add: %d\n", *h_add); // 6
    printf("result multiply: %d\n", *h_mul); // 8

    hipFree(d_result_add);
    hipFree(d_result_multiply);

    return 0;
}